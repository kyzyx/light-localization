#include "hip/hip_runtime.h"
#include "cudamap.h"
#include <cuda_gl_interop.h>
#include "hip/hip_vector_types.h"
#include <stdio.h>

#define BLOCK_SIZE 512
#define MAX_FLOAT 1e9

__device__ static float2 cmpVI(float2 a, float2 b) {
    return a.x<b.x?a:b;
}
__device__ static unsigned long long int _float2_ll(float2 a) {
    return *((unsigned long long int*) &a);
}
__device__ static float2 _ll_float2(unsigned long long int a) {
    return *((float2*) &a);
}

// From http://stackoverflow.com/questions/17399119/cant-we-use-atomic-operations-for-floating-point-variables-in-cuda
__device__ static float2 atomicMin2(float2* address, float2 val)
{
    unsigned long long int* address_as_i = (unsigned long long int*) address;
    unsigned long long int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
                _float2_ll(cmpVI(val, _ll_float2(assumed)))
                );
    } while (assumed != old);
    return _ll_float2(old);
}

__global__ void cuAddlight(
        float* intensities,
        float3* surfel_pos,
        float3* surfel_normal,
        float intensity, float x, float y, float z, int n)
{
    int tid = threadIdx.x;
    int surfaceIdx = tid + blockDim.x*blockIdx.x;

    if (surfaceIdx < n) {
        float3 pos = surfel_pos[surfaceIdx];
        float3 norm = surfel_normal[surfaceIdx];
        float3 p = make_float3(x,y,z);
        float3 L = p - pos;
        float LdotL = dot(L,L);
        float ndotL = dot(norm, L);

        float ret = LdotL>0?ndotL*intensity/(LdotL*sqrt(LdotL)):0;
        atomicAdd(intensities+surfaceIdx, ret);
    }
}

template <unsigned int blockSize>
__global__ void cuCompute(
        float* intensities,
        float3* surfel_pos,
        float3* surfel_normal,
        float3 plane_normal,
        float3 plane_axis,
        float3 plane_point,
        int n,
        float2* field,
        int w, int h
        )
{
    __shared__ float2 mini[BLOCK_SIZE];

    int tid = threadIdx.x;
    int surfaceIdx = tid + blockDim.x*blockIdx.x;
    mini[tid] = make_float2(MAX_FLOAT, 0);

    if (surfaceIdx < n) {
        // Data load
        float intensity = intensities[surfaceIdx];
        float3 pos = surfel_pos[surfaceIdx];
        float3 norm = surfel_normal[surfaceIdx];

        mini[tid].y = __int_as_float(surfaceIdx);

        // Computation
        float3 axis2 = cross(plane_normal, plane_axis);
        float2 pix = make_float2(blockIdx.y/(float)blockDim.y, blockIdx.z/(float)blockDim.z);
        float3 p = pix.x*plane_axis + pix.y*axis2;
        float3 L = p - pos;
        float LdotL = dot(L,L);
        float ndotLn = dot(norm, L)/sqrt(LdotL);
        char occl = 1;
        float v = intensity*occl*ndotLn>0?intensity*LdotL/ndotLn:MAX_FLOAT;
        mini[tid].x = v>0.f?v:MAX_FLOAT;
    }
    __syncthreads();

    // Reduction
    if (blockSize >= 512) {
        if (tid < 256) { mini[tid] = cmpVI(mini[tid+256], mini[tid]); }
        __syncthreads(); 
    }
    if (blockSize >= 256) {
        if (tid < 128) { mini[tid] = cmpVI(mini[tid+128], mini[tid]); }
        __syncthreads(); 
    }
    if (blockSize >= 128) {
        if (tid < 64)  { mini[tid] = cmpVI(mini[tid+64], mini[tid]); }
        __syncthreads(); 
    }
    if (blockSize >= 64)  {
        if (tid < 32)  { mini[tid] = cmpVI(mini[tid+32], mini[tid]); }
        __syncthreads(); 
    }
    if (blockSize >= 32)  {
        if (tid < 16)  { mini[tid] = cmpVI(mini[tid+16], mini[tid]); }
        __syncthreads(); 
    }
    if (blockSize >= 16)  {
        if (tid < 8)   { mini[tid] = cmpVI(mini[tid+8], mini[tid]); }
        __syncthreads(); 
    }
    if (blockSize >= 8)   {
        if (tid < 4)   { mini[tid] = cmpVI(mini[tid+4], mini[tid]); }
        __syncthreads(); 
    }
    if (blockSize >= 4)   {
        if (tid < 2)   { mini[tid] = cmpVI(mini[tid+2], mini[tid]); }
        __syncthreads(); 
    }
    if (blockSize >= 2)   {
        if (tid < 1)   { mini[tid] = cmpVI(mini[tid+1], mini[tid]); }
        __syncthreads(); 
    }

    // Final data copy
    if (tid == 0) {
        atomicMin2(field+blockIdx.z*w+blockIdx.y, mini[0]);
    }
}
void Cudamap_init(Cudamap* cudamap, const float* surfel_pos, const float* surfel_normal) {
    hipSetDevice(0);
    hipMalloc((void**) &(cudamap->d_intensities), sizeof(float)*cudamap->n);
    hipMalloc((void**) &(cudamap->d_surfel_pos), sizeof(float3)*cudamap->n);
    hipMalloc((void**) &(cudamap->d_surfel_normal), sizeof(float3)*cudamap->n);
    hipMalloc((void**) &(cudamap->d_field), sizeof(float2)*cudamap->w*cudamap->h);

    hipMemcpy(cudamap->d_surfel_pos, surfel_pos, sizeof(float3)*cudamap->n, hipMemcpyHostToDevice);
    hipMemcpy(cudamap->d_surfel_normal, surfel_normal, sizeof(float3)*cudamap->n, hipMemcpyHostToDevice);
    hipMemset((void*) cudamap->d_intensities, 0, sizeof(float)*cudamap->n);
}

void Cudamap_setGLTexture(Cudamap* cudamap, unsigned int tex) {
    cudaGLSetGLDevice(0);
    hipStream_t hip_stream;
    hipGraphicsResource *resources[1];

    hipGraphicsGLRegisterImage(resources, tex, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore);
    hipStreamCreate(&hip_stream);
    hipGraphicsMapResources(1, resources, hip_stream);
    hipGraphicsSubResourceGetMappedArray(&(cudamap->d_field_tex), resources[0], 0, 0);
    hipGraphicsUnmapResources(1, resources, hip_stream);
    hipStreamDestroy(hip_stream);
}

void Cudamap_setGLBuffer(Cudamap* cudamap, unsigned int pbo) {
    hipStream_t hip_stream;
    hipGraphicsResource *resources[1];
    cudaGLSetGLDevice(0);
    size_t size;

    hipGraphicsGLRegisterBuffer(resources, pbo, cudaGraphicsMapFlagsNone);
    hipStreamCreate(&hip_stream);
    hipGraphicsMapResources(1, resources, hip_stream);
    hipGraphicsResourceGetMappedPointer((void **)&(cudamap->d_field), &size, resources[0]);
    hipGraphicsUnmapResources(1, resources, hip_stream);
    hipStreamDestroy(hip_stream);
}
void Cudamap_free(Cudamap* cudamap) {
    hipFree(cudamap->d_surfel_pos);
    hipFree(cudamap->d_surfel_normal);
    hipFree(cudamap->d_intensities);
    hipFree(cudamap->d_field);
}
void Cudamap_setIntensities(Cudamap* cudamap, float* intensities) {
    if (intensities) {
        hipMemcpy(cudamap->d_intensities, intensities, sizeof(float)*cudamap->n, hipMemcpyHostToDevice);
    } else {
        hipMemset((void*) cudamap->d_intensities, 0, sizeof(float)*cudamap->n);
    }
}
void Cudamap_addLight(Cudamap* cudamap, float intensity, float x, float y, float z) {
    cuAddlight<<< (cudamap->n+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE >>>(
            cudamap->d_intensities,
            cudamap->d_surfel_pos,
            cudamap->d_surfel_normal,
            intensity, x, y, z, cudamap->n);
}

void Cudamap_compute(Cudamap* cudamap, float* field, const float* plane_normal, const float* plane_axis, const float* plane_point)
{
    static int running = 0;
    int n = cudamap->n;
    int w = cudamap->w;
    int h = cudamap->h;

    if (running) return;
    running = 1;
    for (int i = 0; i < w*h; i++) {
        field[2*i] = MAX_FLOAT;
        field[2*i+1] = 0;
    }
    hipMemcpy(cudamap->d_field, field, sizeof(float2)*w*h, hipMemcpyHostToDevice);

    dim3 threads(BLOCK_SIZE, 1, 1);
    dim3 blocks((n+BLOCK_SIZE-1)/BLOCK_SIZE, w, h);

    cuCompute<BLOCK_SIZE><<< blocks, threads >>>(
            cudamap->d_intensities,
            cudamap->d_surfel_pos,
            cudamap->d_surfel_normal,
            make_float3(plane_normal[0], plane_normal[1], plane_normal[2]),
            make_float3(plane_axis[0], plane_axis[1], plane_axis[2]),
            make_float3(plane_point[0], plane_point[1], plane_point[2]),
            n, cudamap->d_field, w, h
            );

    if (cudamap->d_field_tex) {
        hipMemcpyToArray(cudamap->d_field_tex, 0, 0, cudamap->d_field, sizeof(float2)*w*h, hipMemcpyDeviceToDevice);
    }
    hipMemcpy(field, cudamap->d_field, sizeof(float2)*w*h, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    running = 0;
}
